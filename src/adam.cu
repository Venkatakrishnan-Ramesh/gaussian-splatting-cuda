#include "hip/hip_runtime.h"
// Copyright (c) 2023 Janusch Patas.
#include "adam.cuh"
optim::Adam::Adam(float learning_rate, float beta1, float beta2, float epsilon) : _pos_lr(learning_rate),
                                                                                  _scaling_lr(learning_rate),
                                                                                  _rotation_lr(learning_rate),
                                                                                  _opacity_lr(learning_rate),
                                                                                  _features_dc_lr(learning_rate),
                                                                                  _features_rest_lr(learning_rate),
                                                                                  _beta1(beta1),
                                                                                  _beta2(beta2),
                                                                                  _epsilon(epsilon),
                                                                                  _beta1_t(beta1),
                                                                                  _beta2_t(beta2) {

    hipStreamCreate(&_stream_pos);
    hipStreamCreate(&_stream_scaling);
    hipStreamCreate(&_stream_rotation);
    hipStreamCreate(&_stream_opacity);
    hipStreamCreate(&_stream_features_dc);
    hipStreamCreate(&_stream_features_rest);
}
optim::Adam::~Adam() {
    sync();
    hipStreamDestroy(_stream_pos);
    hipStreamDestroy(_stream_scaling);
    hipStreamDestroy(_stream_rotation);
    hipStreamDestroy(_stream_opacity);
    hipStreamDestroy(_stream_features_dc);
    hipStreamDestroy(_stream_features_rest);

    hipFree(_d_ma_pos);
    hipFree(_d_ma_scaling);
    hipFree(_d_ma_rotation);
    hipFree(_d_ma_opacity);
    hipFree(_d_ma_features_dc);
    hipFree(_d_ma_features_rest);
}
void optim::Adam::InitializePos(std::vector<int> shape, float learning_rate) {
    _pos_lr = learning_rate;

    hipMallocAsync(&_d_ma_pos, sizeof(float) * shape[0] * shape[1], _stream_pos);
    hipMemsetAsync(_d_ma_pos, 0, sizeof(float) * shape[0] * shape[1], _stream_pos);
}
void optim::Adam::InitializeScaling(std::vector<int> shape, float learning_rate) {
    _scaling_lr = learning_rate;

    hipMallocAsync(&_d_ma_scaling, sizeof(float) * shape[0] * shape[1], _stream_scaling);
    hipMemsetAsync(_d_ma_scaling, 0, sizeof(float) * shape[0] * shape[1], _stream_scaling);
}
void optim::Adam::InitializeRotation(std::vector<int> shape, float learning_rate) {
    _rotation_lr = learning_rate;

    hipMallocAsync(&_d_ma_rotation, sizeof(float) * shape[0] * shape[1], _stream_rotation);
    hipMemsetAsync(_d_ma_rotation, 0, sizeof(float) * shape[0] * shape[1], _stream_rotation);
}
void optim::Adam::InitializeOpacity(std::vector<int> shape, float learning_rate) {
    _opacity_lr = learning_rate;

    hipMallocAsync(&_d_ma_opacity, sizeof(float) * shape[0], _stream_opacity);
    hipMemsetAsync(_d_ma_opacity, 0, sizeof(float) * shape[0], _stream_opacity);
}
void optim::Adam::InitializeFeaturesDC(std::vector<int> shape, float learning_rate) {
    _features_dc_lr = learning_rate;

    hipMallocAsync(&_d_ma_features_dc, sizeof(float) * shape[0] * shape[1] * shape[2], _stream_features_dc);
    hipMemsetAsync(_d_ma_features_dc, 0, sizeof(float) * shape[0] * shape[1] * shape[2], _stream_features_dc);
}
void optim::Adam::InitializeFeaturesRest(std::vector<int> shape, float learning_rate) {
    _features_rest_lr = learning_rate;

    hipMallocAsync(&_d_ma_features_rest, sizeof(float) * shape[0] * shape[1] * shape[2], _stream_features_rest);
    hipMemsetAsync(_d_ma_features_rest, 0, sizeof(float) * shape[0] * shape[1] * shape[2], _stream_features_rest);
}

void optim::Adam::step() {
}

void optim::Adam::sync() {
    hipStreamSynchronize(_stream_pos);
    hipStreamSynchronize(_stream_scaling);
    hipStreamSynchronize(_stream_rotation);
    hipStreamSynchronize(_stream_opacity);
    hipStreamSynchronize(_stream_features_dc);
    hipStreamSynchronize(_stream_features_rest);
}

__global__ void AdamUpdateKernel(float* params, float* d_params, float* m, float* v, int size, float lr_t, float beta1, float beta2, float epsilon) {
    // calculate the index for the weight/bias
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // only execute if the index is within the size of the weights/biases
    if (idx < size) {
        // compute the new moving average of the gradient
        m[idx] = beta1 * m[idx] + (1 - beta1) * d_params[idx];

        // compute the new moving average of the squared gradient
        v[idx] = beta2 * v[idx] + (1 - beta2) * d_params[idx] * d_params[idx];

        // update the weights/biases
        params[idx] -= lr_t * m[idx] / (sqrt(v[idx]) + epsilon);
    }
}
