#include "hip/hip_runtime.h"
// Copyright (c) 2023 Janusch Patas.
#include "adam.cuh"
#include "debug_utils.cuh"
#include <utility>

namespace gs {
    namespace optim {

        __global__ void AdamUpdatePos_Scaling_Kernel(float3* params,
                                                     const float3* d_params_grad,
                                                     float3* d_avg,
                                                     float3* d_avg_sq,
                                                     int32_t* __restrict__ d_steps,
                                                     int size, float lr_t,
                                                     float beta1,
                                                     float beta2,
                                                     float epsilon);
        __global__ void AdamUpdateRotationKernel(float4* params,
                                                 const float4* d_params_grad,
                                                 float4* d_avg,
                                                 float4* d_avg_sq,
                                                 int32_t* __restrict__ d_steps,
                                                 int size,
                                                 float lr_t,
                                                 float beta1,
                                                 float beta2,
                                                 float epsilon);
        __global__ void AdamUpdateOpactiyKernel(float* params,
                                                const float* d_params_grad,
                                                float* d_avg,
                                                float* d_avg_sq,
                                                int32_t* __restrict__ d_steps,
                                                int size,
                                                float lr_t,
                                                float beta1,
                                                float beta2,
                                                float epsilon);
        __global__ void AdamUpdateFeatureKernel(float3* params,
                                                const float3* d_params_grad,
                                                float3* d_avg,
                                                float3* d_avg_sq,
                                                int32_t* __restrict__ d_steps,
                                                int size,
                                                int dim1,
                                                float lr_t,
                                                float beta1,
                                                float beta2,
                                                float epsilon);

        AdamParameter::AdamParameter(ParamType param_type,
                                     torch::Tensor param,
                                     float learning_rate,
                                     hipStream_t stream,
                                     float beta1 /*= 0.9f */,
                                     float beta2 /*= 0.999f */,
                                     float epsilon /*= 1e-8 */) : _param_type(param_type),
                                                                  _d_params(param),
                                                                  _param_name(Map_param_type_to_string(param_type)),
                                                                  _lr(learning_rate),
                                                                  _beta1(beta1),
                                                                  _beta2(beta2),
                                                                  _epsilon(epsilon) {
            _d_params_grad = torch::zeros_like(_d_params);
            _d_avg = torch::zeros_like(_d_params);
            _d_avg_sq = torch::zeros_like(_d_params);
            _d_steps = torch::zeros({_d_params.size(0), 1}, torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA));
        }

        AdamParameter::~AdamParameter() {
        }

        void AdamParameter::Step(hipStream_t stream) {
            const bool avg_equal = _d_avg.sizes() == _d_params.sizes();
            const bool param_equal = _d_params.sizes() == _d_params_grad.sizes();
            // This has all be satified to update the paramaters successfully
            //                std::cout << "\nAdamUpdatePos_Scaling_Kernel " + Map_param_type_to_string(GetType()) + " shape: " << _d_params.size(0) << ", " << _d_params.size(1) << std::endl;
            //                std::cout << std::setprecision(6) << "lr " << _lr << ", beta1 " << _beta1 << ", beta2 " << _beta2 << std::endl;
            // Ensure all tensors are on the same device as params
            _d_params_grad = _d_params_grad.to(_d_params.device());
            _d_avg = _d_avg.to(_d_params_grad.device());
            _d_avg = _d_avg.to(_d_params_grad.device());

            // Update biased first and second moment estimates
            _d_avg = _beta1 * _d_avg + (1.f - _beta1) * _d_params_grad;
            _d_avg_sq = _beta2 * _d_avg_sq + (1.f - _beta2) * _d_params_grad * _d_params_grad;

            _d_params -= _lr * _d_avg / (torch::sqrt(_d_avg_sq) + _epsilon);
            // Update parameters
        }

        void Adam::Step(hipStream_t stream) {
            //            for (auto& [key, param] : _params) {
            //                param->Step(stream);
            //                hipDeviceSynchronize();
            //            }
            _params[ParamType::Pos]->Step(stream);
            _params[ParamType::Scaling]->Step(stream);
            _params[ParamType::Rotation]->Step(stream);
            _params[ParamType::Opacity]->Step(stream);
            _params[ParamType::Features_dc]->Step(stream);
            _params[ParamType::Features_rest]->Step(stream);
        }

        void AdamParameter::Set_Exp_Avg_Sq(torch::Tensor d_avg_sq) {
            _d_avg_sq = d_avg_sq;
        }
        void AdamParameter::Set_Exp_Avg(torch::Tensor d_avg) {
            _d_avg = d_avg;
        }

        void AdamParameter::Set_Gradient(torch::Tensor d_param_grad) {
            _d_params_grad = d_param_grad;
        }

        void Adam::AddParameter(std::shared_ptr<AdamParameterBase> param) {
            _params[param->GetType()] = param;
        }

        __global__ void AdamUpdatePos_Scaling_Kernel(float3* __restrict__ params,
                                                     const float3* __restrict__ d_params_grad,
                                                     float3* __restrict__ d_avg,
                                                     float3* __restrict__ d_avg_sq,
                                                     int32_t* __restrict__ d_steps,
                                                     int size,
                                                     float lr_t,
                                                     float beta1,
                                                     float beta2,
                                                     float epsilon) {
            // calculate the index for the weight/bias
            int idx = blockIdx.x * blockDim.x + threadIdx.x;

            // only execute if the index is within the size of the weights/biases
            if (idx < size) {
                // compute the new moving average of the gradient
                float3 avg = d_avg[idx];
                float3 avg_sq = d_avg_sq[idx];
                float3 param = params[idx];
                const float3 param_grad = d_params_grad[idx];
                const int32_t current_step = ++d_steps[idx];

                // Bias correction terms
                float bias_correction1 = 1.0f - powf(beta1, current_step);
                float bias_correction2 = 1.0f - powf(beta2, current_step);
                float bias_correction2_sqrt = sqrtf(bias_correction2);

                avg.x = beta1 * avg.x + (1.f - beta1) * param_grad.x;
                avg.y = beta1 * avg.y + (1.f - beta1) * param_grad.y;
                avg.z = beta1 * avg.z + (1.f - beta1) * param_grad.z;

                // compute the new moving average of the squared gradient
                avg_sq.x = beta2 * avg_sq.x + (1.f - beta2) * param_grad.x * param_grad.x;
                avg_sq.y = beta2 * avg_sq.y + (1.f - beta2) * param_grad.y * param_grad.y;
                avg_sq.z = beta2 * avg_sq.z + (1.f - beta2) * param_grad.z * param_grad.z;

                // Compute step size considering bias correction
                float step_size = lr_t / bias_correction1;

                // update the weights/biases
                param.x -= step_size * avg.x / (sqrtf(avg_sq.x / bias_correction2_sqrt) + epsilon);
                param.y -= step_size * avg.y / (sqrtf(avg_sq.y / bias_correction2_sqrt) + epsilon);
                param.z -= step_size * avg.z / (sqrtf(avg_sq.z / bias_correction2_sqrt) + epsilon);

                params[idx] = param;
                d_avg[idx] = avg;
                d_avg_sq[idx] = avg_sq;
            }
        }

        __global__ void AdamUpdateRotationKernel(float4* __restrict__ params,
                                                 const float4* __restrict__ d_params_grad,
                                                 float4* __restrict__ d_avg,
                                                 float4* __restrict__ d_avg_sq,
                                                 int32_t* __restrict__ d_steps,
                                                 int size,
                                                 float lr_t,
                                                 float beta1,
                                                 float beta2,
                                                 float epsilon) {

            // calculate the index for the weight/bias
            int idx = blockIdx.x * blockDim.x + threadIdx.x;

            // only execute if the index is within the size of the weights/biases
            if (idx < size) {
                float4 avg = d_avg[idx];
                float4 avg_sq = d_avg_sq[idx];
                float4 param = params[idx];
                const float4 param_grad = d_params_grad[idx];
                const int32_t current_step = ++d_steps[idx];

                // Bias correction terms
                float bias_correction1 = 1.0f - powf(beta1, current_step);
                float bias_correction2 = 1.0f - powf(beta2, current_step);
                float bias_correction2_sqrt = sqrtf(bias_correction2);

                avg.x = beta1 * avg.x + (1.f - beta1) * param_grad.x;
                avg.y = beta1 * avg.y + (1.f - beta1) * param_grad.y;
                avg.z = beta1 * avg.z + (1.f - beta1) * param_grad.z;
                avg.w = beta1 * avg.w + (1.f - beta1) * param_grad.w;

                // compute the new moving average of the squared gradient
                avg_sq.x = beta2 * avg_sq.x + (1.f - beta2) * param_grad.x * param_grad.x;
                avg_sq.y = beta2 * avg_sq.y + (1.f - beta2) * param_grad.y * param_grad.y;
                avg_sq.z = beta2 * avg_sq.z + (1.f - beta2) * param_grad.z * param_grad.z;
                avg_sq.w = beta2 * avg_sq.w + (1.f - beta2) * param_grad.w * param_grad.w;

                // Compute step size considering bias correction
                float step_size = lr_t / bias_correction1;

                // update the weights/biases
                param.x -= step_size * avg.x / (sqrtf(avg_sq.x / bias_correction2_sqrt) + epsilon);
                param.y -= step_size * avg.y / (sqrtf(avg_sq.y / bias_correction2_sqrt) + epsilon);
                param.z -= step_size * avg.z / (sqrtf(avg_sq.z / bias_correction2_sqrt) + epsilon);
                param.w -= step_size * avg.w / (sqrtf(avg_sq.w / bias_correction2_sqrt) + epsilon);
                params[idx] = param;
                d_avg[idx] = avg;
                d_avg_sq[idx] = avg_sq;
            }
        }

        __global__ void AdamUpdateOpactiyKernel(float* params,
                                                const float* d_params_grad,
                                                float* d_avg,
                                                float* d_avg_sq,
                                                int32_t* __restrict__ d_steps,
                                                int size,
                                                float lr_t,
                                                float beta1,
                                                float beta2,
                                                float epsilon) {
            // calculate the index for the weight/bias
            int idx = blockIdx.x * blockDim.x + threadIdx.x;

            // only execute if the index is within the size of the weights/biases
            if (idx < size) {
                // compute the new moving average of the gradient
                float avg = d_avg[idx];
                float avg_sq = d_avg_sq[idx];
                float param = params[idx];
                const float param_grad = d_params_grad[idx];
                const int32_t current_step = ++d_steps[idx];

                // Bias correction terms
                float bias_correction1 = 1.0f - powf(beta1, current_step);
                float bias_correction2 = 1.0f - powf(beta2, current_step);
                float bias_correction2_sqrt = sqrtf(bias_correction2);

                avg = beta1 * avg + (1.f - beta1) * param_grad;

                // compute the new moving average of the squared gradient
                avg_sq = beta2 * avg_sq + (1.f - beta2) * param_grad * param_grad;

                float step_size = lr_t / bias_correction1;

                // update the weights/biases
                param -= step_size * avg / (sqrtf(avg_sq / bias_correction2_sqrt) + epsilon);

                params[idx] = param;
                d_avg[idx] = avg;
                d_avg_sq[idx] = avg_sq;
            }
        }

        __global__ void AdamUpdateFeatureKernel(float3* params,
                                                const float3* d_params_grad,
                                                float3* d_avg,
                                                float3* d_avg_sq,
                                                int32_t* __restrict__ d_steps,
                                                int size,
                                                int dim1,
                                                float lr_t,
                                                float beta1,
                                                float beta2,
                                                float epsilon) {
            // calculate the index for the weight/bias
            int idx = blockIdx.x * blockDim.x + threadIdx.x;

            // only execute if the index is within the size of the weights/biases
            if (idx < size) {
                const int32_t current_step = ++d_steps[idx];
                // Bias correction terms
                float bias_correction1 = 1.0f - powf(beta1, current_step);
                float bias_correction2 = 1.0f - powf(beta2, current_step);
                float bias_correction2_sqrt = sqrtf(bias_correction2);

                for (int j = 0; j < dim1; j++) {
                    const int current_index = idx * dim1 + j;
                    float3 avg = d_avg[current_index];
                    float3 avg_sq = d_avg_sq[current_index];
                    float3 param = params[current_index];
                    const float3 param_grad = d_params_grad[current_index];

                    avg.x = beta1 * avg.x + (1.f - beta1) * param_grad.x;
                    avg.y = beta1 * avg.y + (1.f - beta1) * param_grad.y;
                    avg.z = beta1 * avg.z + (1.f - beta1) * param_grad.z;

                    // compute the new moving average of the squared gradient
                    avg_sq.x = beta2 * avg_sq.x + (1.f - beta2) * param_grad.x * param_grad.x;
                    avg_sq.y = beta2 * avg_sq.y + (1.f - beta2) * param_grad.y * param_grad.y;
                    avg_sq.z = beta2 * avg_sq.z + (1.f - beta2) * param_grad.z * param_grad.z;

                    // Compute step size considering bias correction
                    float step_size = lr_t / bias_correction1;

                    // update the weights/biases
                    param.x -= step_size * avg.x / (sqrtf(avg_sq.x / bias_correction2_sqrt) + epsilon);
                    param.y -= step_size * avg.y / (sqrtf(avg_sq.y / bias_correction2_sqrt) + epsilon);
                    param.z -= step_size * avg.z / (sqrtf(avg_sq.z / bias_correction2_sqrt) + epsilon);
                    params[current_index] = param;
                    d_avg[current_index] = avg;
                    d_avg_sq[current_index] = avg_sq;
                }
            }
        }
    } // namespace optim
} // namespace gs