#include "hip/hip_runtime.h"
#include "debug_utils.cuh"
#include "gaussian.cuh"
#include "read_utils.cuh"
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_runtime.h>
#include <exception>
#include <memory>
#include <thread>

GaussianModel::GaussianModel(int sh_degree) : _max_sh_degree(sh_degree) {
    hipStreamCreate(&_stream1);
    hipStreamCreate(&_stream2);
    hipStreamCreate(&_stream3);
    hipStreamCreate(&_stream4);
    hipStreamCreate(&_stream5);
    hipStreamCreate(&_stream6);
}

GaussianModel::~GaussianModel() {
    hipStreamDestroy(_stream1);
    hipStreamDestroy(_stream2);
    hipStreamDestroy(_stream3);
    hipStreamDestroy(_stream4);
    hipStreamDestroy(_stream5);
    hipStreamDestroy(_stream6);
}

/**
 * @brief Fetches the features of the Gaussian model
 *
 * This function concatenates _features_dc and _features_rest along the second dimension.
 *
 * @return Tensor of the concatenated features
 */
torch::Tensor GaussianModel::Get_features() const {
    auto features_dc = _features_dc;
    auto features_rest = _features_rest;
    return torch::cat({features_dc, features_rest}, 1);
}

/**
 * @brief Increment the SH degree by 1
 *
 * This function increments the active_sh_degree by 1, up to a maximum of max_sh_degree.
 */
void GaussianModel::One_up_sh_degree() {
    if (_active_sh_degree < _max_sh_degree) {
        _active_sh_degree++;
    }
}

/**
 * @brief Initialize Gaussian Model from a Point Cloud.
 *
 * This function creates a Gaussian model from a given PointCloud object. It also sets
 * the spatial learning rate scale. The model's features, scales, rotations, and opacities
 * are initialized based on the input point cloud.
 *
 * @param pcd The input point cloud
 * @param spatial_lr_scale The spatial learning rate scale
 */
void GaussianModel::Create_from_pcd(PointCloud& pcd, float spatial_lr_scale) {
    _spatial_lr_scale = spatial_lr_scale;

    const auto pointType = torch::TensorOptions().dtype(torch::kFloat32);
    _xyz = torch::from_blob(pcd._points.data(), {static_cast<long>(pcd._points.size()), 3}, pointType).to(torch::kCUDA);
    auto dist2 = torch::clamp_min(distCUDA2(_xyz), 0.0000001);
    _scaling = torch::log(torch::sqrt(dist2)).unsqueeze(-1).repeat({1, 3}).to(torch::kCUDA, true);
    _rotation = torch::zeros({_xyz.size(0), 4}).index_put_({torch::indexing::Slice(), 0}, 1).to(torch::kCUDA, true);
    _opacity = inverse_sigmoid(0.5 * torch::ones({_xyz.size(0), 1})).to(torch::kCUDA, true);
    _max_radii2D = torch::zeros({_xyz.size(0)}).to(torch::kCUDA, true);

    // colors
    auto colorType = torch::TensorOptions().dtype(torch::kUInt8);
    auto fused_color = RGB2SH(torch::from_blob(pcd._colors.data(), {static_cast<long>(pcd._colors.size()), 3}, colorType).to(pointType) / 255.f).to(torch::kCUDA);

    // features
    auto features = torch::zeros({fused_color.size(0), 3, static_cast<long>(std::pow((_max_sh_degree + 1), 2))}).to(torch::kCUDA);
    features.index_put_({torch::indexing::Slice(), torch::indexing::Slice(torch::indexing::None, 3), 0}, fused_color);
    features.index_put_({torch::indexing::Slice(), torch::indexing::Slice(3, torch::indexing::None), torch::indexing::Slice(1, torch::indexing::None)}, 0.0);
    _features_dc = features.index({torch::indexing::Slice(), torch::indexing::Slice(), torch::indexing::Slice(0, 1)}).transpose(1, 2).contiguous();
    _features_rest = features.index({torch::indexing::Slice(), torch::indexing::Slice(), torch::indexing::Slice(1, torch::indexing::None)}).transpose(1, 2).contiguous();
}

/**
 * @brief Setup the Gaussian Model for training
 *
 * This function sets up the Gaussian model for training by initializing several
 * parameters and settings based on the provided OptimizationParameters object.
 *
 * @param params The OptimizationParameters object providing the settings for training
 */
void GaussianModel::Training_setup(const OptimizationParameters& params) {
    this->_percent_dense = params.percent_dense;
    this->_xyz_gradient_accum = torch::zeros({this->_xyz.size(0), 1}).to(torch::kCUDA);
    this->_denom = torch::zeros({this->_xyz.size(0), 1}).to(torch::kCUDA);
    this->_xyz_scheduler_args = Expon_lr_func(params.position_lr_init * this->_spatial_lr_scale,
                                              params.position_lr_final * this->_spatial_lr_scale,
                                              params.position_lr_delay_mult,
                                              params.position_lr_max_steps);

    const std::vector<int> xyz_shape = {static_cast<int>(_xyz.size(0)), static_cast<int>(_xyz.size(1))};
    const std::vector<int> features_dc_shape = {static_cast<int>(_features_dc.size(0)), static_cast<int>(_features_dc.size(1)), static_cast<int>(_features_dc.size(2))};
    const std::vector<int> features_rest_shape = {static_cast<int>(_features_rest.size(0)), static_cast<int>(_features_rest.size(1)), static_cast<int>(_features_rest.size(2))};
    const std::vector<int> scaling_shape = {static_cast<int>(_scaling.size(0)), static_cast<int>(_scaling.size(1))};
    const std::vector<int> rotation_shape = {static_cast<int>(_rotation.size(0)), static_cast<int>(_rotation.size(1))};
    const std::vector<int> opacity_shape = {static_cast<int>(_opacity.size(0)), static_cast<int>(_opacity.size(1))};

    _optimizer = std::make_unique<gs::optim::Adam>();
    _optimizer->AddParameter(std::make_shared<gs::optim::AdamParameter<gs::optim::pos_param_t>>(gs::optim::ParamType::Pos,
                                                                                                xyz_shape,
                                                                                                params.position_lr_init * this->_spatial_lr_scale));
    _optimizer->AddParameter(std::make_shared<gs::optim::AdamParameter<gs::optim::feature_dc_param_t>>(gs::optim::ParamType::Features_dc,
                                                                                                       features_dc_shape,
                                                                                                       params.feature_lr));
    _optimizer->AddParameter(std::make_shared<gs::optim::AdamParameter<gs::optim::feature_rest_param_t>>(gs::optim::ParamType::Features_rest,
                                                                                                         features_rest_shape,
                                                                                                         params.feature_lr / 20.f));
    _optimizer->AddParameter(std::make_shared<gs::optim::AdamParameter<gs::optim::scaling_param_t>>(gs::optim::ParamType::Scaling,
                                                                                                    scaling_shape,
                                                                                                    params.scaling_lr * this->_spatial_lr_scale));
    _optimizer->AddParameter(std::make_shared<gs::optim::AdamParameter<gs::optim::rotation_param_t>>(gs::optim::ParamType::Rotation,
                                                                                                     rotation_shape,
                                                                                                     params.rotation_lr));
    _optimizer->AddParameter(std::make_shared<gs::optim::AdamParameter<gs::optim::opacity_param_t>>(gs::optim::ParamType::Opacity,
                                                                                                    opacity_shape,
                                                                                                    params.opacity_lr));

    _optimizer->Sync();
}

void GaussianModel::Update_learning_rate(float iteration) {
    // This is hacky because you cant change in libtorch individual parameter learning rate
    // xyz is added first, since _optimizer->param_groups() return a vector, we assume that xyz stays first
    auto lr = _xyz_scheduler_args(iteration);
    _optimizer->GetParameters(gs::optim::ParamType::Pos)->UpdateLearningRate(lr);
}

void GaussianModel::Update_Params_and_Grads(const torch::Tensor& grad_means3D,
                                            const torch::Tensor& grad_sh, // needs to be splitted
                                            const torch::Tensor& grad_opacities,
                                            const torch::Tensor& grad_scales,
                                            const torch::Tensor& grad_rotations) {
    auto xyz = reinterpret_cast<gs::optim::pos_param_t*>(_xyz.data_ptr<float>());
    auto features_dc = reinterpret_cast<gs::optim::feature_dc_param_t*>(_features_dc.data_ptr<float>());
    auto features_rest = reinterpret_cast<gs::optim::feature_rest_param_t*>(_features_rest.data_ptr<float>());
    auto scaling = reinterpret_cast<gs::optim::scaling_param_t*>(_scaling.data_ptr<float>());
    auto rotation = reinterpret_cast<gs::optim::rotation_param_t*>(_rotation.data_ptr<float>());
    auto opacity = reinterpret_cast<gs::optim::opacity_param_t*>(_opacity.data_ptr<float>());

    _optimizer->GetAdamParameter<gs::optim::pos_param_t>(gs::optim::ParamType::Pos)->Update_Parameter_Pointer(xyz);
    _optimizer->GetAdamParameter<gs::optim::feature_dc_param_t>(gs::optim::ParamType::Features_dc)->Update_Parameter_Pointer(features_dc);
    _optimizer->GetAdamParameter<gs::optim::feature_rest_param_t>(gs::optim::ParamType::Features_rest)->Update_Parameter_Pointer(features_rest);
    _optimizer->GetAdamParameter<gs::optim::scaling_param_t>(gs::optim::ParamType::Scaling)->Update_Parameter_Pointer(scaling);
    _optimizer->GetAdamParameter<gs::optim::rotation_param_t>(gs::optim::ParamType::Rotation)->Update_Parameter_Pointer(rotation);
    _optimizer->GetAdamParameter<gs::optim::opacity_param_t>(gs::optim::ParamType::Opacity)->Update_Parameter_Pointer(opacity);

    auto grad_features_dc = grad_sh.index({torch::indexing::Slice(), torch::indexing::Slice(0, 1), torch::indexing::Slice()}).contiguous();
    auto grad_features_rest = grad_sh.index({torch::indexing::Slice(), torch::indexing::Slice(1, torch::indexing::None), torch::indexing::Slice()}).contiguous();

    _optimizer->GetAdamParameter<gs::optim::pos_param_t>(gs::optim::ParamType::Pos)->Set_Gradient(reinterpret_cast<gs::optim::pos_param_t*>(grad_means3D.data_ptr<float>()), {static_cast<int>(grad_means3D.size(0), 3)});
    _optimizer->GetAdamParameter<gs::optim::feature_dc_param_t>(gs::optim::ParamType::Features_dc)->Set_Gradient(reinterpret_cast<gs::optim::feature_dc_param_t*>(grad_features_dc.data_ptr<float>()), {static_cast<int>(grad_features_dc.size(0)), static_cast<int>(grad_features_dc.size(1)), static_cast<int>(grad_features_dc.size(2))});
    _optimizer->GetAdamParameter<gs::optim::feature_rest_param_t>(gs::optim::ParamType::Features_rest)->Set_Gradient(reinterpret_cast<gs::optim::feature_rest_param_t*>(grad_features_rest.data_ptr<float>()), {static_cast<int>(grad_features_rest.size(0)), static_cast<int>(grad_features_rest.size(1)), static_cast<int>(grad_features_rest.size(2))});
    _optimizer->GetAdamParameter<gs::optim::scaling_param_t>(gs::optim::ParamType::Scaling)->Set_Gradient(reinterpret_cast<gs::optim::scaling_param_t*>(grad_scales.data_ptr<float>()), {static_cast<int>(grad_scales.size(0)), static_cast<int>(grad_scales.size(1))});
    _optimizer->GetAdamParameter<gs::optim::rotation_param_t>(gs::optim::ParamType::Rotation)->Set_Gradient(reinterpret_cast<gs::optim::rotation_param_t*>(grad_rotations.data_ptr<float>()), {static_cast<int>(grad_rotations.size(0)), static_cast<int>(grad_rotations.size(1))});
    _optimizer->GetAdamParameter<gs::optim::opacity_param_t>(gs::optim::ParamType::Opacity)->Set_Gradient(reinterpret_cast<gs::optim::opacity_param_t*>(grad_opacities.data_ptr<float>()), {static_cast<int>(grad_opacities.size(0)), static_cast<int>(grad_opacities.size(1))});
}

void GaussianModel::Reset_opacity() {
    // opacitiy activation
    _opacity = inverse_sigmoid(torch::ones_like(_opacity, torch::TensorOptions().dtype(torch::kFloat32)) * 0.01f);
    auto updateTensor = torch::zeros_like(_opacity);
    // new optimizer
    auto opacity_params = _optimizer->GetAdamParameter<gs::optim::opacity_param_t>(gs::optim::ParamType::Opacity);
    opacity_params->Set_Exp_Avg(reinterpret_cast<gs::optim::opacity_param_t*>(updateTensor.data_ptr<float>()), {static_cast<int>(_opacity.size(0))});
    opacity_params->Set_Exp_Avg_Sq(reinterpret_cast<gs::optim::opacity_param_t*>(updateTensor.data_ptr<float>()), {static_cast<int>(_opacity.size(0))});
}

void copy3DAsync(const float* src,
                 const std::vector<long>& src_size,
                 float* dst,
                 const std::vector<long>& dst_size,
                 hipStream_t stream) {
    hipMemcpy3DParms copyParams = {0};
    copyParams.kind = hipMemcpyDeviceToDevice;

    copyParams.srcPtr = make_hipPitchedPtr(
        (void*)src,
        (size_t)src_size[2] * sizeof(float),
        (size_t)src_size[2],
        (size_t)src_size[1]);

    copyParams.dstPtr = make_hipPitchedPtr(
        (void*)dst,
        (size_t)dst_size[2] * sizeof(float),
        (size_t)dst_size[2],
        (size_t)dst_size[1]);

    copyParams.extent = make_hipExtent(
        (size_t)src_size[2] * sizeof(float),
        (size_t)src_size[1],
        (size_t)src_size[0]);
    CHECK_CUDA_ERROR(hipMemcpy3DAsync(&copyParams, stream));
}

void copy2DAsync(const float* src, const std::vector<long>& src_size, float* dst, hipStream_t stream) {
    //    float *new_dst;
    //    hipMalloc(&new_dst, dst.size(0) * dst.size(1) * sizeof(float));
    if (src_size.size() != 2) {
        return;
    }
    size_t width_in_bytes = src_size[1] * sizeof(float);
    size_t height_in_elements = src_size[0];
    size_t src_pitch = src_size[1] * sizeof(float); // provide stride
    size_t dst_pitch = src_size[1] * sizeof(float); // make stride

    hipMemcpy2DAsync(
        dst,                      // Destination pointer
        dst_pitch,                // Destination pitch
        src,                      // Source pointer
        src_pitch,                // Source pitch
        width_in_bytes,           // Width of the 2D region in bytes
        height_in_elements,       // Height of the 2D region in elements
        hipMemcpyDeviceToDevice, // Specifies the kind of copy (Device to Device)
        stream                    // Stream to perform the copy
    );
    CHECK_LAST_CUDA_ERROR();
}

template <typename T>
__global__ void index_select_2D_kernel(const T* __restrict__ A,
                                       T* __restrict__ B,
                                       const long* __restrict__ indices,
                                       int size_B) {
    const int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= size_B) {
        return;
    }

    B[row] = A[indices[row]];
}

template <typename T>
__global__ void index_select_3D_kernel(const T* __restrict__ A,
                                       T* __restrict__ B,
                                       const long* __restrict__ indices,
                                       int size_B,
                                       int dim1) {
    const int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= size_B) {
        return;
    }

    for (int i = 0; i < dim1; ++i) {
        B[row * dim1 + i] = A[indices[row] * dim1 + i];
    }
}

template <typename param_t>
void prune_optimizer(const torch::Tensor& mask,
                     torch::Tensor& old_tensor,
                     gs::optim::Adam* new_optimizer,
                     gs::optim::ParamType param_type,
                     const std::vector<int>& shape) {

    const auto options = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA);
    const int threads = 1024;
    const int blocks = (mask.size(0) + threads - 1) / threads;

    hipStream_t stream1;
    hipStream_t stream2;
    hipStream_t stream3;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);
    switch (param_type) {
    case gs::optim::ParamType::Pos:
    case gs::optim::ParamType::Scaling:
    case gs::optim::ParamType::Rotation:
    case gs::optim::ParamType::Opacity: {
        auto masked_param = torch::zeros({mask.size(0), shape[1]}, options);
        auto exp_avg = torch::zeros({mask.size(0), shape[1]}, options);
        auto exp_avg_sq = torch::zeros({mask.size(0), shape[1]}, options);
        auto param = new_optimizer->GetAdamParameter<param_t>(param_type);
        index_select_2D_kernel<param_t><<<blocks, threads, 0, stream1>>>(reinterpret_cast<param_t*>(old_tensor.data_ptr<float>()),
                                                                         reinterpret_cast<param_t*>(masked_param.data_ptr<float>()),
                                                                         mask.data_ptr<long>(),
                                                                         static_cast<int>(mask.size(0)));

        index_select_2D_kernel<param_t><<<blocks, threads, 0, stream2>>>(param->Get_Exp_Avg(),
                                                                         reinterpret_cast<param_t*>(exp_avg.data_ptr<float>()),
                                                                         mask.data_ptr<long>(),
                                                                         static_cast<int>(mask.size(0)));
        index_select_2D_kernel<param_t><<<blocks, threads, 0, stream3>>>(param->Get_Exp_Avg_Sq(),
                                                                         reinterpret_cast<param_t*>(exp_avg_sq.data_ptr<float>()),
                                                                         mask.data_ptr<long>(),
                                                                         static_cast<int>(mask.size(0)));
        hipStreamSynchronize(stream1);
        hipStreamSynchronize(stream2);
        hipStreamSynchronize(stream3);
        param->Set_Exp_Avg(reinterpret_cast<param_t*>(exp_avg_sq.data_ptr<float>()), shape);
        param->Set_Exp_Avg_Sq(reinterpret_cast<param_t*>(exp_avg_sq.data_ptr<float>()), shape);
        old_tensor = masked_param;
    } break;
    case gs::optim::ParamType::Features_dc:
    case gs::optim::ParamType::Features_rest: {
        auto masked_param = torch::zeros({mask.size(0), shape[1], shape[2]}, options);
        auto exp_avg = torch::zeros({mask.size(0), shape[1], shape[2]}, options);
        auto exp_avg_sq = torch::zeros({mask.size(0), shape[1], shape[2]}, options);
        auto param = new_optimizer->GetAdamParameter<param_t>(param_type);

        index_select_3D_kernel<param_t><<<blocks, threads, 0, stream1>>>(reinterpret_cast<param_t*>(old_tensor.data_ptr<float>()),
                                                                         reinterpret_cast<param_t*>(masked_param.data_ptr<float>()),
                                                                         mask.data_ptr<long>(),
                                                                         static_cast<int>(mask.size(0)),
                                                                         static_cast<int>(old_tensor.size(1)));

        index_select_3D_kernel<param_t><<<blocks, threads, 0, stream2>>>(param->Get_Exp_Avg(),
                                                                         reinterpret_cast<param_t*>(exp_avg.data_ptr<float>()),
                                                                         mask.data_ptr<long>(),
                                                                         static_cast<int>(mask.size(0)),
                                                                         static_cast<int>(old_tensor.size(1)));
        index_select_3D_kernel<param_t><<<blocks, threads, 0, stream3>>>(param->Get_Exp_Avg_Sq(),
                                                                         reinterpret_cast<param_t*>(exp_avg_sq.data_ptr<float>()),
                                                                         mask.data_ptr<long>(),
                                                                         static_cast<int>(mask.size(0)),
                                                                         static_cast<int>(old_tensor.size(1)));

        hipStreamSynchronize(stream1);
        hipStreamSynchronize(stream2);
        hipStreamSynchronize(stream3);
        param->Set_Exp_Avg(reinterpret_cast<param_t*>(exp_avg_sq.data_ptr<float>()), shape);
        param->Set_Exp_Avg_Sq(reinterpret_cast<param_t*>(exp_avg_sq.data_ptr<float>()), shape);
        old_tensor = masked_param;
    } break;
    default:
        throw std::runtime_error("Not implemented cast in tensors_to_optimizer_new");
    }

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(stream3);
}

void GaussianModel::prune_points(torch::Tensor mask) {
    // reverse to keep points
    auto valid_point_mask = ~mask;
    auto indices = torch::nonzero(valid_point_mask == true).index({torch::indexing::Slice(torch::indexing::None, torch::indexing::None), torch::indexing::Slice(torch::indexing::None, 1)}).squeeze(-1);
    prune_optimizer<gs::optim::pos_param_t>(indices, _xyz, _optimizer.get(), gs::optim::ParamType::Pos, {static_cast<int>(_xyz.size(0)), static_cast<int>(_xyz.size(1))});
    prune_optimizer<gs::optim::feature_dc_param_t>(indices, _features_dc, _optimizer.get(), gs::optim::ParamType::Features_dc, {static_cast<int>(_features_dc.size(0)), static_cast<int>(_features_dc.size(1)), static_cast<int>(_features_dc.size(2))});
    prune_optimizer<gs::optim::feature_rest_param_t>(indices, _features_rest, _optimizer.get(), gs::optim::ParamType::Features_rest, {static_cast<int>(_features_rest.size(0)), static_cast<int>(_features_rest.size(1)), static_cast<int>(_features_rest.size(2))});
    prune_optimizer<gs::optim::scaling_param_t>(indices, _scaling, _optimizer.get(), gs::optim::ParamType::Scaling, {static_cast<int>(_scaling.size(0)), static_cast<int>(_scaling.size(1))});
    prune_optimizer<gs::optim::rotation_param_t>(indices, _rotation, _optimizer.get(), gs::optim::ParamType::Rotation, {static_cast<int>(_rotation.size(0)), static_cast<int>(_rotation.size(1))});
    prune_optimizer<gs::optim::opacity_param_t>(indices, _opacity, _optimizer.get(), gs::optim::ParamType::Opacity, {static_cast<int>(_opacity.size(0)), static_cast<int>(_opacity.size(1))});

    _xyz_gradient_accum = _xyz_gradient_accum.index_select(0, indices);
    _denom = _denom.index_select(0, indices);
    _max_radii2D = _max_radii2D.index_select(0, indices);
}

template <typename param_t>
void tensors_to_optimizer_new(torch::Tensor& extended_tensor,
                              torch::Tensor& old_tensor,
                              torch::Tensor& exp_avg,
                              torch::Tensor exp_avg_sq,
                              gs::optim::Adam* new_optimizer,
                              gs::optim::ParamType param_type) {

    // new optimizer
    std::vector<int> shape;
    for (int i = 0; i < extended_tensor.sizes().size(); ++i) {
        shape.push_back(extended_tensor.size(i));
    }

    auto param = new_optimizer->GetAdamParameter<param_t>(param_type);
    param->Set_Exp_Avg(reinterpret_cast<param_t*>(exp_avg_sq.data_ptr<float>()), shape);
    param->Set_Exp_Avg_Sq(reinterpret_cast<param_t*>(exp_avg_sq.data_ptr<float>()), shape);
    old_tensor = extended_tensor;
}

template <typename param_t>
void cat_tensors_to_optimizer(torch::Tensor& extension_tensor,
                              torch::Tensor& old_tensor,
                              gs::optim::Adam* new_optimizer,
                              gs::optim::ParamType param_type) {
    hipStream_t _stream1;
    hipStreamCreate(&_stream1);

    auto new_exp_avg = torch::tensor({});
    auto new_exp_avg_sq = torch::tensor({});
    const auto options = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA);

    auto param = new_optimizer->GetAdamParameter<param_t>(param_type);

    torch::Tensor new_tensor;
    if (param_type == gs::optim::ParamType::Features_rest || param_type == gs::optim::ParamType::Features_dc) {
        std::vector<long> old_shape = {static_cast<int>(old_tensor.size(0)),
                                       static_cast<int>(old_tensor.size(1)),
                                       static_cast<int>(old_tensor.size(2))};
        std::vector<long> extension_shape = {static_cast<int>(extension_tensor.size(0)),
                                             static_cast<int>(extension_tensor.size(1)),
                                             static_cast<int>(extension_tensor.size(2))};
        std::vector<long> new_shape = old_shape;
        new_shape[0] += extension_tensor.size(0);

        new_exp_avg = torch::zeros({new_shape[0], new_shape[1], new_shape[2]}, options);
        new_exp_avg_sq = torch::zeros({new_shape[0], new_shape[1], new_shape[2]}, options);
        // TODO need to copy form new optimizer
        copy3DAsync(reinterpret_cast<float*>(param->Get_Exp_Avg()), old_shape, new_exp_avg.data_ptr<float>(), new_shape, _stream1);
        copy3DAsync(reinterpret_cast<float*>(param->Get_Exp_Avg_Sq()), old_shape, new_exp_avg_sq.data_ptr<float>(), new_shape, _stream1);

        new_tensor = torch::zeros({new_shape[0], new_shape[1], new_shape[2]}, options);
        copy3DAsync(old_tensor.data_ptr<float>(), old_shape, new_tensor.data_ptr<float>(), new_shape, _stream1);
        auto shifted_ptr = new_tensor.data_ptr<float>() + old_shape[0] * old_shape[1] * old_shape[2];
        copy3DAsync(extension_tensor.data_ptr<float>(), extension_shape, shifted_ptr, extension_shape, _stream1);

        // TODO: somehow slipped in. Verfiy that it works
        old_tensor = torch::cat({old_tensor, extension_tensor}, 0);
    } else {
        std::vector<long> old_shape = {static_cast<int>(old_tensor.size(0)), static_cast<int>(old_tensor.size(1))};
        std::vector<long> new_shape = old_shape;
        std::vector<long> extension_shape = {static_cast<int>(extension_tensor.size(0)), static_cast<int>(extension_tensor.size(1))};
        new_shape[0] += extension_tensor.size(0);

        new_exp_avg = torch::zeros({new_shape[0], new_shape[1]}, options);
        new_exp_avg_sq = torch::zeros({new_shape[0], new_shape[1]}, options);

        // TODO: eed to copy form new optimizer
        copy2DAsync(reinterpret_cast<float*>(param->Get_Exp_Avg()), old_shape, new_exp_avg.data_ptr<float>(), _stream1);
        copy2DAsync(reinterpret_cast<float*>(param->Get_Exp_Avg_Sq()), old_shape, new_exp_avg_sq.data_ptr<float>(), _stream1);

        new_tensor = torch::zeros({new_shape[0], new_shape[1]}, options);
        copy2DAsync(old_tensor.data_ptr<float>(), old_shape, new_tensor.data_ptr<float>(), _stream1);
        auto shifted_ptr = new_tensor.data_ptr<float>() + old_shape[0] * old_shape[1];
        copy2DAsync(extension_tensor.data_ptr<float>(), extension_shape, shifted_ptr, _stream1);
        old_tensor = new_tensor;
    }

    hipStreamSynchronize(_stream1);
    hipStreamDestroy(_stream1);

    std::vector<int> shape;
    for (int i = 0; i < old_tensor.sizes().size(); ++i) {
        shape.push_back(old_tensor.size(i));
    }

    param->Set_Exp_Avg(reinterpret_cast<param_t*>(new_exp_avg_sq.data_ptr<float>()), shape);
    param->Set_Exp_Avg_Sq(reinterpret_cast<param_t*>(new_exp_avg_sq.data_ptr<float>()), shape);
}

void GaussianModel::densification_postfix(torch::Tensor& new_xyz,
                                          torch::Tensor& new_features_dc,
                                          torch::Tensor& new_features_rest,
                                          torch::Tensor& new_scaling,
                                          torch::Tensor& new_rotation,
                                          torch::Tensor& new_opacity) {
    cat_tensors_to_optimizer<gs::optim::pos_param_t>(new_xyz, _xyz, _optimizer.get(), gs::optim::ParamType::Pos);
    cat_tensors_to_optimizer<gs::optim::feature_dc_param_t>(new_features_dc, _features_dc, _optimizer.get(), gs::optim::ParamType::Features_dc);
    cat_tensors_to_optimizer<gs::optim::feature_rest_param_t>(new_features_rest, _features_rest, _optimizer.get(), gs::optim::ParamType::Features_rest);
    cat_tensors_to_optimizer<gs::optim::scaling_param_t>(new_scaling, _scaling, _optimizer.get(), gs::optim::ParamType::Scaling);
    cat_tensors_to_optimizer<gs::optim::rotation_param_t>(new_rotation, _rotation, _optimizer.get(), gs::optim::ParamType::Rotation);
    cat_tensors_to_optimizer<gs::optim::opacity_param_t>(new_opacity, _opacity, _optimizer.get(), gs::optim::ParamType::Opacity);

    _xyz_gradient_accum = torch::zeros({_xyz.size(0), 1}).to(torch::kCUDA);
    _denom = torch::zeros({_xyz.size(0), 1}).to(torch::kCUDA);
    _max_radii2D = torch::zeros({_xyz.size(0)}).to(torch::kCUDA);
}

void GaussianModel::densify_and_split(torch::Tensor& grads, float grad_threshold, float scene_extent, float min_opacity, float max_screen_size) {
    static const int N = 2;
    const int n_init_points = _xyz.size(0);
    // Extract points that satisfy the gradient condition
    torch::Tensor padded_grad = torch::zeros({n_init_points}).to(torch::kCUDA);
    padded_grad.slice(0, 0, grads.size(0)) = grads.squeeze();
    torch::Tensor selected_pts_mask = torch::where(padded_grad >= grad_threshold, torch::ones_like(padded_grad).to(torch::kBool), torch::zeros_like(padded_grad).to(torch::kBool));
    selected_pts_mask = torch::logical_and(selected_pts_mask, std::get<0>(Get_scaling().max(1)) > _percent_dense * scene_extent);
    auto indices = torch::nonzero(selected_pts_mask.squeeze(-1) == true).index({torch::indexing::Slice(torch::indexing::None, torch::indexing::None), torch::indexing::Slice(torch::indexing::None, 1)}).squeeze(-1);

    torch::Tensor stds = Get_scaling().index_select(0, indices).repeat({N, 1});
    torch::Tensor means = torch::zeros({stds.size(0), 3}).to(torch::kCUDA);
    torch::Tensor samples = torch::randn({stds.size(0), stds.size(1)}).to(torch::kCUDA) * stds + means;
    torch::Tensor rots = build_rotation(_rotation.index_select(0, indices)).repeat({N, 1, 1});

    torch::Tensor new_xyz = torch::bmm(rots, samples.unsqueeze(-1)).squeeze(-1) + _xyz.index_select(0, indices).repeat({N, 1});
    torch::Tensor new_scaling = torch::log(Get_scaling().index_select(0, indices).repeat({N, 1}) / (0.8 * N));
    torch::Tensor new_rotation = _rotation.index_select(0, indices).repeat({N, 1});
    // TODO I strongly belief that this split is totaly unneccessary. Just makes overhead.
    torch::Tensor new_features_dc = _features_dc.index_select(0, indices).repeat({N, 1, 1});
    torch::Tensor new_features_rest = _features_rest.index_select(0, indices).repeat({N, 1, 1});
    torch::Tensor new_opacity = _opacity.index_select(0, indices).repeat({N, 1});

    densification_postfix(new_xyz, new_features_dc, new_features_rest, new_scaling, new_rotation, new_opacity);

    torch::Tensor prune_filter = torch::cat({selected_pts_mask.squeeze(-1), torch::zeros({N * selected_pts_mask.sum().item<int>()}).to(torch::kBool).to(torch::kCUDA)});
    // torch::Tensor prune_filter = torch::cat({selected_pts_mask.squeeze(-1), torch::zeros({N * selected_pts_mask.sum().item<int>()})}).to(torch::kBool).to(torch::kCUDA);
    prune_filter = torch::logical_or(prune_filter, (Get_opacity() < min_opacity).squeeze(-1));
    prune_points(prune_filter);
}

__global__ void concat_selection_float3_kernel(
    const float3* __restrict__ src,
    const int64_t* __restrict__ indices,
    float3* __restrict__ dst,
    const size_t extension_size,
    const size_t orig_size) {

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= extension_size) {
        return;
    }

    const int64_t src_index = indices[idx];
    const int64_t dest_idx = orig_size + idx;

    // Single memory read operation for each 3D point
    dst[dest_idx] = src[src_index];
}

__global__ void concat_selection_float4_kernel(
    const float4* __restrict__ src,
    const int64_t* __restrict__ indices,
    float4* __restrict__ dst,
    const size_t extension_size,
    const size_t orig_size) {

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= extension_size) {
        return;
    }

    const int64_t src_index = indices[idx];
    const int64_t dest_idx = orig_size + idx;

    // Single memory read operation for each 3D point
    dst[dest_idx] = src[src_index];
}

__global__ void concat_elements_kernel_opacity(
    const float* __restrict__ opacity,
    const int64_t* __restrict__ indices,
    float* __restrict__ new_opacity,
    const size_t N,
    const size_t orig_N) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) {
        return;
    }

    const int64_t index = indices[idx];
    const int64_t dest_idx = orig_N + idx;
    new_opacity[dest_idx] = opacity[index];
}

__global__ void concat_elements_kernel_features_dc(
    const float3* __restrict__ features_dc,
    const int64_t* __restrict__ indices,
    float3* __restrict__ new_features_dc,
    const size_t N,
    const size_t orig_N,
    const size_t F1) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) {
        return;
    }

    // Copy selected elements to new tensors, at positions after the original elements
    const int64_t index = indices[idx];
    const int64_t dest_idx = orig_N + idx;

    for (int j = 0; j < F1; j++) {
        new_features_dc[dest_idx * F1 + j] = features_dc[index * F1 + j];
    }
}

__global__ void concat_elements_kernel_features_rest(
    const float3* __restrict__ features_rest,
    const int64_t* __restrict__ indices,
    float3* __restrict__ new_features_rest,
    const size_t N,
    const size_t orig_N,
    const size_t F1) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) {
        return;
    }

    // Copy selected elements to new tensors, at positions after the original elements
    const int64_t index = indices[idx];
    const int64_t dest_idx = orig_N + idx;

    for (int j = 0; j < F1; j++) {
        new_features_rest[dest_idx * F1 + j] = features_rest[index * F1 + j];
    }
}

void copy1DAsync(const torch::Tensor& src, torch::Tensor& dst, hipStream_t stream) {
    // assert(src.size(0) <= dst.size(0));

    size_t bytes_to_copy = src.size(0) * sizeof(float);

    CHECK_CUDA_ERROR(hipMemcpyAsync(
        dst.data_ptr(),           // Destination pointer
        src.data_ptr(),           // Source pointer
        bytes_to_copy,            // Number of bytes to copy
        hipMemcpyDeviceToDevice, // Specifies the kind of copy (Device to Device)
        stream                    // Stream to perform the copy
        ));
}

void GaussianModel::select_elements_and_cat(
    const float* xyz,
    const std::vector<long>& xyz_size,
    const float* features_dc,
    const std::vector<long>& features_dc_size,
    const float* features_rest,
    const std::vector<long>& features_rest_size,
    const float* opacity,
    const std::vector<long>& opacity_size,
    const float* scaling,
    const std::vector<long>& scaling_size,
    const float* rotation,
    const std::vector<long>& rotation_size,
    int64_t* indices,
    long original_size,
    long extension_size) {

    long threads = 256;
    long blocks = std::max(1L, (extension_size + threads - 1L) / threads);
    {

        auto xyz_paramstates = _optimizer->GetAdamParameter<gs::optim::pos_param_t>(gs::optim::ParamType::Pos);
        auto features_dc_paramstates = _optimizer->GetAdamParameter<gs::optim::feature_dc_param_t>(gs::optim::ParamType::Features_dc);
        auto features_rest_paramstates = _optimizer->GetAdamParameter<gs::optim::feature_rest_param_t>(gs::optim::ParamType::Features_rest);
        auto scaling_paramstates = _optimizer->GetAdamParameter<gs::optim::scaling_param_t>(gs::optim::ParamType::Scaling);
        auto rotation_paramstates = _optimizer->GetAdamParameter<gs::optim::rotation_param_t>(gs::optim::ParamType::Rotation);
        auto opacity_paramstates = _optimizer->GetAdamParameter<gs::optim::opacity_param_t>(gs::optim::ParamType::Opacity);

        const auto options = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA);
        auto new_xyz = torch::zeros({_xyz.size(0) + extension_size, _xyz.size(1)}, options);
        auto new_features_dc = torch::zeros({original_size + extension_size, _features_dc.size(1), _features_dc.size(2)}, options);
        auto new_features_rest = torch::zeros({original_size + extension_size, _features_rest.size(1), _features_rest.size(2)}, options);
        auto new_opacity = torch::zeros({original_size + extension_size, 1}, options);
        auto new_scaling = torch::zeros({original_size + extension_size, 3}, options);
        auto new_rotation = torch::zeros({original_size + extension_size, 4}, options);

        const int total_extension_count = original_size + extension_size;
        auto xyz_exp_avg = torch::zeros({total_extension_count, 3}, options);
        auto xyz_exp_avg_sq = torch::zeros({total_extension_count, 3}, options);
        auto features_dc_avg = torch::zeros({total_extension_count, _features_dc.size(1), _features_dc.size(2)}, options);
        auto features_dc_avg_sq = torch::zeros({total_extension_count, _features_dc.size(1), _features_dc.size(2)}, options);
        auto features_rest_avg = torch::zeros({total_extension_count, _features_rest.size(1), _features_rest.size(2)}, options);
        auto features_rest_avg_sq = torch::zeros({total_extension_count, _features_rest.size(1), _features_rest.size(2)}, options);
        auto opacity_avg = torch::zeros({total_extension_count, _opacity.size(1)}, options);
        auto opacity_avg_sq = torch::zeros({total_extension_count, _opacity.size(1)}, options);
        auto scaling_avg = torch::zeros({total_extension_count, _scaling.size(1)}, options);
        auto scaling_avg_sq = torch::zeros({total_extension_count, _scaling.size(1)}, options);
        auto rotation_avg = torch::zeros({total_extension_count, _rotation.size(1)}, options);
        auto rotation_avg_sq = torch::zeros({total_extension_count, _rotation.size(1)}, options);

        // TODO: no reinterpret_cast -> refactor accordingly. Should be also faster if we take the native floatx type?
        copy2DAsync(reinterpret_cast<float*>(xyz_paramstates->Get_Exp_Avg()), {original_size, 3}, xyz_exp_avg.data_ptr<float>(), _stream1);
        copy2DAsync(reinterpret_cast<float*>(xyz_paramstates->Get_Exp_Avg_Sq()), {original_size, 3}, xyz_exp_avg_sq.data_ptr<float>(), _stream1);

        copy3DAsync(reinterpret_cast<float*>(features_dc_paramstates->Get_Exp_Avg()), features_dc_size, features_dc_avg.data_ptr<float>(), {original_size + extension_size, features_dc_size[1], features_dc_size[2]}, _stream2);
        copy3DAsync(reinterpret_cast<float*>(features_dc_paramstates->Get_Exp_Avg_Sq()), features_dc_size, features_dc_avg_sq.data_ptr<float>(), {original_size + extension_size, features_dc_size[1], features_dc_size[2]}, _stream2);

        copy3DAsync(reinterpret_cast<float*>(features_rest_paramstates->Get_Exp_Avg()), features_rest_size, features_rest_avg.data_ptr<float>(), {original_size + extension_size, features_rest_size[1], features_rest_size[2]}, _stream3);
        copy3DAsync(reinterpret_cast<float*>(features_rest_paramstates->Get_Exp_Avg_Sq()), features_rest_size, features_rest_avg_sq.data_ptr<float>(), {original_size + extension_size, features_rest_size[1], features_rest_size[2]}, _stream3);

        // opacity is already float*
        copy2DAsync(opacity_paramstates->Get_Exp_Avg(), {original_size, 1}, opacity_avg.data_ptr<float>(), _stream4);
        copy2DAsync(opacity_paramstates->Get_Exp_Avg_Sq(), {original_size, 1}, opacity_avg_sq.data_ptr<float>(), _stream4);

        copy2DAsync(reinterpret_cast<float*>(scaling_paramstates->Get_Exp_Avg()), {original_size, 3}, scaling_avg.data_ptr<float>(), _stream5);
        copy2DAsync(reinterpret_cast<float*>(scaling_paramstates->Get_Exp_Avg_Sq()), {original_size, 3}, scaling_avg_sq.data_ptr<float>(), _stream5);

        copy2DAsync(reinterpret_cast<float*>(rotation_paramstates->Get_Exp_Avg()), {original_size, 4}, rotation_avg.data_ptr<float>(), _stream6);
        copy2DAsync(reinterpret_cast<float*>(rotation_paramstates->Get_Exp_Avg_Sq()), {original_size, 4}, rotation_avg_sq.data_ptr<float>(), _stream6);

        copy2DAsync(xyz, xyz_size, new_xyz.data_ptr<float>(), _stream1);
        const auto* xyz3_ptr = reinterpret_cast<const float3*>(xyz);
        auto* new_xyz3_ptr = reinterpret_cast<float3*>(new_xyz.data_ptr<float>());
        concat_selection_float3_kernel<<<blocks, threads, 0, _stream1>>>(
            xyz3_ptr,
            indices,
            new_xyz3_ptr,
            extension_size,
            xyz_size[0]);

        copy2DAsync(scaling, scaling_size, new_scaling.data_ptr<float>(), _stream5);
        const auto* scaling3_ptr = reinterpret_cast<const float3*>(scaling);
        auto* new_scaling3_ptr = reinterpret_cast<float3*>(new_scaling.data_ptr<float>());
        concat_selection_float3_kernel<<<blocks, threads, 0, _stream5>>>(
            scaling3_ptr,
            indices,
            new_scaling3_ptr,
            extension_size,
            scaling_size[0]);
        copy3DAsync(features_dc, features_dc_size, new_features_dc.data_ptr<float>(), {original_size + extension_size, features_dc_size[1], features_dc_size[2]}, _stream2);
        const auto* features_dc_ptr = reinterpret_cast<const float3*>(features_dc);
        auto* new_features_dc_ptr = reinterpret_cast<float3*>(new_features_dc.data_ptr<float>());
        concat_elements_kernel_features_dc<<<blocks, threads, 0, _stream2>>>(
            features_dc_ptr,
            indices,
            new_features_dc_ptr,
            extension_size,
            xyz_size[0], features_dc_size[1]);
        copy2DAsync(opacity, opacity_size, new_opacity.data_ptr<float>(), _stream4);
        concat_elements_kernel_opacity<<<blocks, threads, 0, _stream4>>>(
            opacity,
            indices,
            new_opacity.data_ptr<float>(),
            extension_size,
            xyz_size[0]);
        copy3DAsync(features_rest, features_rest_size, new_features_rest.data_ptr<float>(), {original_size + extension_size, features_rest_size[1], features_rest_size[2]}, _stream3);
        const auto* features_rest_ptr = reinterpret_cast<const float3*>(features_rest);
        auto* new_features_rest_ptr = reinterpret_cast<float3*>(new_features_rest.data_ptr<float>());
        concat_elements_kernel_features_rest<<<blocks, threads, 0, _stream3>>>(
            features_rest_ptr,
            indices,
            new_features_rest_ptr,
            extension_size,
            features_rest_size[0], features_rest_size[1]);
        CHECK_LAST_CUDA_ERROR();

        copy2DAsync(rotation, rotation_size, new_rotation.data_ptr<float>(), _stream6);
        auto* rotation_ptr = reinterpret_cast<const float4*>(rotation);
        auto* new_rotation_ptr = reinterpret_cast<float4*>(new_rotation.data_ptr<float>());
        concat_selection_float4_kernel<<<blocks, threads, 0, _stream6>>>(
            rotation_ptr,
            indices,
            new_rotation_ptr,
            extension_size,
            rotation_size[0]);
        CHECK_LAST_CUDA_ERROR();

        hipStreamSynchronize(_stream1);
        CHECK_LAST_CUDA_ERROR();
        hipStreamSynchronize(_stream2);
        CHECK_LAST_CUDA_ERROR();
        hipStreamSynchronize(_stream3);
        CHECK_LAST_CUDA_ERROR();
        hipStreamSynchronize(_stream4);
        CHECK_LAST_CUDA_ERROR();
        hipStreamSynchronize(_stream5);
        CHECK_LAST_CUDA_ERROR();
        hipStreamSynchronize(_stream6);

        tensors_to_optimizer_new<gs::optim::pos_param_t>(new_xyz, _xyz, xyz_exp_avg, xyz_exp_avg_sq, _optimizer.get(), gs::optim::ParamType::Pos);
        tensors_to_optimizer_new<gs::optim::feature_dc_param_t>(new_features_dc, _features_dc, features_dc_avg, features_dc_avg_sq, _optimizer.get(), gs::optim::ParamType::Features_dc);
        tensors_to_optimizer_new<gs::optim::feature_rest_param_t>(new_features_rest, _features_rest, features_rest_avg, features_rest_avg_sq, _optimizer.get(), gs::optim::ParamType::Features_rest);
        tensors_to_optimizer_new<gs::optim::opacity_param_t>(new_opacity, _opacity, opacity_avg, opacity_avg_sq, _optimizer.get(), gs::optim::ParamType::Opacity);
        tensors_to_optimizer_new<gs::optim::scaling_param_t>(new_scaling, _scaling, scaling_avg, scaling_avg_sq, _optimizer.get(), gs::optim::ParamType::Scaling);
        tensors_to_optimizer_new<gs::optim::rotation_param_t>(new_rotation, _rotation, rotation_avg, rotation_avg_sq, _optimizer.get(), gs::optim::ParamType::Rotation);
    }

    CHECK_LAST_CUDA_ERROR();
}

void GaussianModel::densify_and_clone(torch::Tensor& grads, float grad_threshold, float scene_extent) {
    // Extract points that satisfy the gradient condition
    torch::Tensor selected_pts_mask = torch::where(torch::linalg::vector_norm(grads, {2}, 1, true, torch::kFloat32) >= grad_threshold,
                                                   torch::ones_like(grads.index({torch::indexing::Slice()})).to(torch::kBool),
                                                   torch::zeros_like(grads.index({torch::indexing::Slice()})).to(torch::kBool))
                                          .to(torch::kLong);

    selected_pts_mask = torch::logical_and(selected_pts_mask, std::get<0>(Get_scaling().max(1)).unsqueeze(-1) <= _percent_dense * scene_extent);

    auto indices = torch::nonzero(selected_pts_mask.squeeze(-1) == true).index({torch::indexing::Slice(torch::indexing::None, torch::indexing::None), torch::indexing::Slice(torch::indexing::None, 1)}).squeeze(-1);
    const auto extension_count = torch::sum(selected_pts_mask).item<int>();
    const auto total_extension_count = _xyz.size(0) + extension_count;
    at::cuda::CUDAStream stream1 = at::cuda::getStreamFromPool();
    at::cuda::setCurrentCUDAStream(stream1);
    _xyz_gradient_accum = torch::zeros({total_extension_count, 1}).to(torch::kCUDA);
    _denom = torch::zeros({total_extension_count, 1}).to(torch::kCUDA);
    _max_radii2D = torch::zeros({total_extension_count}).to(torch::kCUDA);
    at::cuda::setCurrentCUDAStream(at::cuda::getDefaultCUDAStream());

    select_elements_and_cat(_xyz.data_ptr<float>(),
                            {_xyz.size(0), _xyz.size(1)},
                            _features_dc.data_ptr<float>(),
                            {_features_dc.size(0), _features_dc.size(1), _features_dc.size(2)},
                            _features_rest.data_ptr<float>(),
                            {_features_rest.size(0), _features_rest.size(1), _features_rest.size(2)},
                            _opacity.data_ptr<float>(),
                            {_opacity.size(0), _opacity.size(1)},
                            _scaling.data_ptr<float>(),
                            {_scaling.size(0), _scaling.size(1)},
                            _rotation.data_ptr<float>(),
                            {_rotation.size(0), _rotation.size(1)},
                            indices.data_ptr<int64_t>(),
                            _xyz.size(0),
                            extension_count);
    //    stream1.synchronize();
}

void GaussianModel::Densify_and_prune(float max_grad, float min_opacity, float extent, float max_screen_size) {
    torch::Tensor grads = _xyz_gradient_accum / _denom;
    grads.index_put_({grads.isnan()}, 0.0);

    densify_and_clone(grads, max_grad, extent);
    densify_and_split(grads, max_grad, extent, min_opacity, max_screen_size);
}

void GaussianModel::Add_densification_stats(torch::Tensor& grad_means2D, torch::Tensor& update_filter) {
    auto filtered_grad = grad_means2D.index_select(0, update_filter.nonzero().squeeze()).slice(1, 0, 2).norm(2, -1, true);
    _xyz_gradient_accum.index_put_({update_filter}, _xyz_gradient_accum.index_select(0, update_filter.nonzero().squeeze()) + filtered_grad);
    _denom.index_put_({update_filter}, _denom.index_select(0, update_filter.nonzero().squeeze()) + 1);
}

std::vector<std::string> GaussianModel::construct_list_of_attributes() {
    std::vector<std::string> attributes = {"x", "y", "z", "nx", "ny", "nz"};

    for (int i = 0; i < _features_dc.size(1) * _features_dc.size(2); ++i)
        attributes.push_back("f_dc_" + std::to_string(i));

    for (int i = 0; i < _features_rest.size(1) * _features_rest.size(2); ++i)
        attributes.push_back("f_rest_" + std::to_string(i));

    attributes.emplace_back("opacity");

    for (int i = 0; i < _scaling.size(1); ++i)
        attributes.push_back("scale_" + std::to_string(i));

    for (int i = 0; i < _rotation.size(1); ++i)
        attributes.push_back("rot_" + std::to_string(i));

    return attributes;
}

void GaussianModel::Save_ply(const std::filesystem::path& file_path, int iteration, bool isLastIteration) {
    auto folder = file_path / ("point_cloud/iteration_" + std::to_string(iteration));
    std::filesystem::create_directories(folder);

    auto xyz = _xyz.cpu().contiguous();
    auto normals = torch::zeros_like(xyz);
    auto f_dc = _features_dc.transpose(1, 2).flatten(1).cpu().contiguous();
    auto f_rest = _features_rest.transpose(1, 2).flatten(1).cpu().contiguous();
    auto opacities = _opacity.cpu();
    auto scale = _scaling.cpu();
    auto rotation = _rotation.cpu();

    std::vector<torch::Tensor> tensor_attributes = {xyz.clone(),
                                                    normals.clone(),
                                                    f_dc.clone(),
                                                    f_rest.clone(),
                                                    opacities.clone(),
                                                    scale.clone(),
                                                    rotation.clone()};
    auto attributes = construct_list_of_attributes();
    std::thread t = std::thread([folder, tensor_attributes, attributes]() {
        Write_output_ply(folder / "point_cloud.ply", tensor_attributes, attributes);
    });

    if (isLastIteration) {
        t.join();
    } else {
        t.detach();
    }
}