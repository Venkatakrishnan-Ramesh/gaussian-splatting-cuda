#include "hip/hip_runtime.h"
#include "debug_utils.cuh"
#include "gaussian.cuh"
#include "loss_monitor.cuh"
#include "loss_utils.cuh"
#include "parameters.cuh"
#include "render_utils.cuh"
#include "scene.cuh"
#include <args.hxx>
#include <filesystem>
#include <fstream>
#include <iostream>
#include <random>
#include <torch/torch.h>

void Write_model_parameters_to_file(const ModelParameters& params) {
    std::filesystem::path outputPath = params.output_path;
    std::filesystem::create_directories(outputPath); // Make sure the directory exists

    std::ofstream cfg_log_f(outputPath / "cfg_args");
    if (!cfg_log_f.is_open()) {
        std::cerr << "Failed to open file for writing!" << std::endl;
        return;
    }

    // Write the parameters in the desired format
    cfg_log_f << "Namespace(";
    cfg_log_f << "eval=" << (params.eval ? "True" : "False") << ", ";
    cfg_log_f << "images='" << params.images << "', ";
    cfg_log_f << "model_path='" << params.output_path.string() << "', ";
    cfg_log_f << "resolution=" << params.resolution << ", ";
    cfg_log_f << "sh_degree=" << params.sh_degree << ", ";
    cfg_log_f << "source_path='" << params.source_path.string() << "', ";
    cfg_log_f << "white_background=" << (params.white_background ? "True" : "False") << ")";
    cfg_log_f.close();

    std::cout << "Output folder: " << params.output_path.string() << std::endl;
}

std::vector<int> get_random_indices(int max_index) {
    std::vector<int> indices(max_index);
    std::iota(indices.begin(), indices.end(), 0);
    // Shuffle the vector
    std::shuffle(indices.begin(), indices.end(), std::default_random_engine());
    return indices;
}

int parse_cmd_line_args(const std::vector<std::string>& args,
                        ModelParameters& modelParams,
                        OptimizationParameters& optimParams,
                        PipelineParameters& pipelineParams) {
    if (args.empty()) {
        std::cerr << "No command line arguments provided!" << std::endl;
        return -1;
    }
    args::ArgumentParser parser("3D Gaussian Splatting CUDA Implementation\n",
                                "This program provides a lightning-fast CUDA implementation of the 3D Gaussian Splatting algorithm for real-time radiance field rendering.");
    args::HelpFlag help(parser, "help", "Display this help menu", {'h', "help"});
    args::ValueFlag<float> convergence_rate(parser, "convergence_rate", "Set convergence rate", {'c', "convergence_rate"});
    args::Flag enable_cr_monitoring(parser, "enable_cr_monitoring", "Enable convergence rate monitoring", {"enable-cr-monitoring"});
    args::ValueFlag<std::string> data_path(parser, "data_path", "Path to the training data", {'d', "data-path"});
    args::ValueFlag<std::string> output_path(parser, "output_path", "Path to the training output", {'o', "output-path"});
    args::ValueFlag<uint32_t> iterations(parser, "iterations", "Number of iterations to train the model", {'i', "iter"});
    args::CompletionFlag completion(parser, {"complete"});

    try {
        parser.Prog(args.front());
        parser.ParseArgs(std::vector<std::string>(args.begin() + 1, args.end()));
    } catch (const args::Completion& e) {
        std::cout << e.what();
        return 0;
    } catch (const args::Help&) {
        std::cout << parser;
        return -1;
    } catch (const args::ParseError& e) {
        std::cerr << e.what() << std::endl;
        std::cerr << parser;
        return -1;
    }

    if (data_path) {
        modelParams.source_path = args::get(data_path);
    } else {
        std::cerr << "No data path provided!" << std::endl;
        return -1;
    }
    if (output_path) {
        modelParams.output_path = args::get(output_path);
    } else {
        std::filesystem::path executablePath = std::filesystem::canonical("/proc/self/exe");
        std::filesystem::path parentDir = executablePath.parent_path().parent_path();
        std::filesystem::path outputDir = parentDir / "output";
        try {
            if (!std::filesystem::create_directory(outputDir)) {
                std::cerr << "Directory already exists! Not overwriting it" << std::endl;
                return -1;
            }
        } catch (...) {
            std::cerr << "Failed to create output directory!" << std::endl;
            return -1;
        }
        modelParams.output_path = outputDir;
    }
    if (iterations) {
        optimParams.iterations = args::get(iterations);
    }
    optimParams.early_stopping = args::get(enable_cr_monitoring);
    if (optimParams.early_stopping && convergence_rate) {
        optimParams.convergence_threshold = args::get(convergence_rate);
    }
    return 0;
}

int main(int argc, char* argv[]) {
    std::vector<std::string> args;
    args.reserve(argc);

    for (int i = 0; i < argc; ++i) {
        args.emplace_back(argv[i]);
    }
    // TODO: read parameters from JSON file or command line
    auto modelParams = ModelParameters();
    auto optimParams = OptimizationParameters();
    auto pipelineParams = PipelineParameters();
    if (parse_cmd_line_args(args, modelParams, optimParams, pipelineParams) < 0) {
        return -1;
    };
    Write_model_parameters_to_file(modelParams);

    auto gaussians = GaussianModel(modelParams.sh_degree);
    auto scene = Scene(gaussians, modelParams);
    gaussians.Training_setup(optimParams);
    if (!torch::cuda::is_available()) {
        // At the moment, I want to make sure that my GPU is utilized.
        std::cout << "CUDA is not available! Training on CPU." << std::endl;
        exit(-1);
    }
    auto pointType = torch::TensorOptions().dtype(torch::kFloat32);
    auto background = modelParams.white_background ? torch::tensor({1.f, 1.f, 1.f}) : torch::tensor({0.f, 0.f, 0.f}, pointType).to(torch::kCUDA);

    const int camera_count = scene.Get_camera_count();
    std::vector<int> indices;
    float loss_add = 0.f;

    LossMonitor loss_monitor(200);
    float avg_converging_rate = 0.f;

    for (int iter = 1; iter < optimParams.iterations + 1; ++iter) {
        if (iter % 1000 == 0) {
            gaussians.One_up_sh_degree();
        }

        if (indices.empty()) {
            indices = get_random_indices(camera_count);
        }
        const int camera_index = indices.back();
        indices.pop_back(); // remove last element to iterate over all cameras randomly
        auto& cam = scene.Get_training_camera(camera_index);
        // Render
        auto [image, viewspace_point_tensor, visibility_filter, radii] = render(cam, gaussians, pipelineParams, background);

        // Loss Computations
        auto gt_image = cam.Get_original_image().to(torch::kCUDA);
        auto l1l = gaussian_splatting::l1_loss(image, gt_image);
        auto loss = (1.f - optimParams.lambda_dssim) * l1l + optimParams.lambda_dssim * (1.f - gaussian_splatting::ssim(image, gt_image));
        //        std::cout << "Iteration: " << iter << " Loss: " << loss.item<float>() << " gaussian splats: " << gaussians.Get_xyz().size(0) << std::endl;
        if (optimParams.early_stopping) {
            avg_converging_rate = loss_monitor.Update(loss.item<float>());
        }
        loss_add += loss.item<float>();
        loss.backward();

        {
            torch::NoGradGuard no_grad;
            auto visible_max_radii = gaussians._max_radii2D.masked_select(visibility_filter);
            auto visible_radii = radii.masked_select(visibility_filter);
            auto max_radii = torch::max(visible_max_radii, visible_radii);
            gaussians._max_radii2D.masked_scatter_(visibility_filter, max_radii);

            if (iter == optimParams.iterations) {
                gaussians.Save_ply(modelParams.output_path, iter, true);
                return 0;
            }

            if (iter % 7'000 == 0) {
                gaussians.Save_ply(modelParams.output_path, iter, false);
            }

            if (iter % 100 == 0) {
                std::cout << "Iteration: " << iter
                          << " Loss: " << loss_add / 100.f
                          << " Average Convergence rate: " << avg_converging_rate
                          << " gaussian splats: " << gaussians.Get_xyz().size(0) << "\n";
                loss_add = 0.f;
            }
            // Densification
            if (iter < optimParams.densify_until_iter) {
                gaussians.Add_densification_stats(viewspace_point_tensor, visibility_filter);
                if (iter > optimParams.densify_from_iter && iter % optimParams.densification_interval == 0) {
                    // @TODO: Not sure about type
                    float size_threshold = iter > optimParams.opacity_reset_interval ? 20.f : -1.f;
                    gaussians.Densify_and_prune(optimParams.densify_grad_threshold, 0.005f, scene.Get_cameras_extent(), size_threshold);
                }

                if (iter % optimParams.opacity_reset_interval == 0 || (modelParams.white_background && iter == optimParams.densify_from_iter)) {
                    std::cout << "iteration " << iter << " resetting opacity" << std::endl;
                    gaussians.Reset_opacity();
                }
            }

            if (iter >= optimParams.densify_until_iter && loss_monitor.IsConverging(optimParams.convergence_threshold)) {
                std::cout << "Converged after " << iter << " iterations!" << std::endl;
                gaussians.Save_ply(modelParams.output_path, iter, true);
                return 0;
            }

            //  Optimizer step
            if (iter < optimParams.iterations) {
                gaussians._optimizer->step();
                gaussians._optimizer->zero_grad(true);
                // @TODO: Not sure about type
                gaussians.Update_learning_rate(iter);
            }
        }
    }
    return 0;
}
