#include "hip/hip_runtime.h"
#include "debug_utils.cuh"
#include "gaussian.cuh"
#include "loss_utils.cuh"
#include "parameters.cuh"
#include "render_utils.cuh"
#include "scene.cuh"
#include <filesystem>
#include <fstream>
#include <iostream>
#include <random>
#include <torch/torch.h>

std::filesystem::path createOutputDirectoryInParent() {
    std::filesystem::path executablePath = std::filesystem::canonical("/proc/self/exe");
    std::filesystem::path parentDir = executablePath.parent_path().parent_path();
    std::filesystem::path outputDir = parentDir / "output";
    std::filesystem::create_directory(outputDir);
    return outputDir;
}

void Write_model_parameters_to_file(const ModelParameters& params) {
    std::filesystem::path outputPath = params.model_path;
    std::filesystem::create_directories(outputPath); // Make sure the directory exists

    std::ofstream cfg_log_f(outputPath / "cfg_args");
    if (!cfg_log_f.is_open()) {
        std::cerr << "Failed to open file for writing!" << std::endl;
        return;
    }

    // Write the parameters in the desired format
    cfg_log_f << "Namespace(";
    cfg_log_f << "eval=" << (params.eval ? "True" : "False") << ", ";
    cfg_log_f << "images='" << params.images << "', ";
    cfg_log_f << "model_path='" << params.model_path.string() << "', ";
    cfg_log_f << "resolution=" << params.resolution << ", ";
    cfg_log_f << "sh_degree=" << params.sh_degree << ", ";
    cfg_log_f << "source_path='" << params.source_path.string() << "', ";
    cfg_log_f << "white_background=" << (params.white_background ? "True" : "False") << ")";
    cfg_log_f.close();

    std::cout << "Output folder: " << params.model_path.string() << std::endl;
}

std::vector<int> get_random_indices(int max_index) {
    std::vector<int> indices(max_index);
    std::iota(indices.begin(), indices.end(), 0);
    // Shuffle the vector
    std::shuffle(indices.begin(), indices.end(), std::default_random_engine());
    return indices;
}

int main(int argc, char* argv[]) {

    if (argc != 2) {
        std::cout << "Usage: ./readPly <ply file>" << std::endl;
        return 1;
    }
    // TODO: read parameters from JSON file or command line
    auto modelParams = ModelParameters();
    modelParams.source_path = argv[1];
    modelParams.model_path = createOutputDirectoryInParent();
    Write_model_parameters_to_file(modelParams);

    const auto optimParams = OptimizationParameters();
    const auto pipelineParams = PipelineParameters();
    auto gaussians = GaussianModel(modelParams.sh_degree);
    auto scene = Scene(gaussians, modelParams);
    gaussians.Training_setup(optimParams);
    if (!torch::cuda::is_available()) {
        // At the moment, I want to make sure that my GPU is utilized.
        std::cout << "CUDA is not available! Training on CPU." << std::endl;
        exit(-1);
    }
    auto pointType = torch::TensorOptions().dtype(torch::kFloat32);
    auto background = modelParams.white_background ? torch::tensor({1.f, 1.f, 1.f}) : torch::tensor({0.f, 0.f, 0.f}, pointType).to(torch::kCUDA);

    const int camera_count = scene.Get_camera_count();
    std::vector<int> indices;
    for (int iter = 1; iter < optimParams.iterations + 1; ++iter) {
        if (iter % 1000 == 0) {
            gaussians.One_up_sh_degree();
        }

        if (indices.empty()) {
            indices = get_random_indices(camera_count);
        }
        const int camera_index = indices.back();
        indices.pop_back(); // remove last element to iterate over all cameras randomly
        auto& cam = scene.Get_training_camera(camera_index);
        // Render
        auto [image, viewspace_point_tensor, visibility_filter, radii] = render(cam, gaussians, pipelineParams, background);

        // Loss Computations
        auto gt_image = cam.Get_original_image().to(torch::kCUDA);
        auto l1l = gaussian_splatting::l1_loss(image, gt_image);
        auto loss = (1.f - optimParams.lambda_dssim) * l1l + optimParams.lambda_dssim * (1.f - gaussian_splatting::ssim(image, gt_image));
        std::cout << "Iteration: " << iter << " Loss: " << loss.item<float>() << " gaussian splats: " << gaussians.Get_xyz().size(0) << std::endl;
        loss.backward();

        {
            torch::NoGradGuard no_grad;
            auto visible_max_radii = gaussians._max_radii2D.masked_select(visibility_filter);
            auto visible_radii = radii.masked_select(visibility_filter);
            auto max_radii = torch::max(visible_max_radii, visible_radii);
            gaussians._max_radii2D.masked_scatter_(visibility_filter, max_radii);

            if (iter % 7'000 == 0) {
                gaussians.Save_ply(modelParams.model_path, iter, true);
                exit(0);
            }
            if (iter == 30'000) {
                gaussians.Save_ply(modelParams.model_path, iter, true);
                return 0;
            }

            // Densification
            if (iter < optimParams.densify_until_iter) {
                gaussians.Add_densification_stats(viewspace_point_tensor, visibility_filter);
                if (iter > optimParams.densify_from_iter && iter % optimParams.densification_interval == 0) {
                    // @TODO: Not sure about type
                    float size_threshold = iter > optimParams.opacity_reset_interval ? 20.f : -1.f;
                    gaussians.Densify_and_prune(optimParams.densify_grad_threshold, 0.005f, scene.Get_cameras_extent(), size_threshold);
                }

                if (iter % optimParams.opacity_reset_interval == 0 || (modelParams.white_background && iter == optimParams.densify_from_iter)) {
                    std::cout << "iteration " << iter << " resetting opacity" << std::endl;
                    gaussians.Reset_opacity();
                }
            }

            //  Optimizer step
            if (iter < optimParams.iterations) {
                gaussians._optimizer->step();
                gaussians._optimizer->zero_grad(true);
                // @TODO: Not sure about type
                gaussians.Update_learning_rate(iter);
            }
        }
    }
    return 0;
}